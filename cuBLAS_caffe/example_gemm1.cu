//Example for using GEMM
#include <stdio.h>
#include <stdlib.h>
//#include <iostream.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

//Matrix is N X M
#define M 3
#define N 3

//#define ij2l(i,j,K) (i*K + j)	
//This used to save in row major format. We need to save in coloumn major format
#define ij2l(i,j,k) (i + j*k)
//This uses coloum major format for saving matrices
//As the im2col implemented in part 3 would be row major, we would have to write our own im2col_colmaj
int main(int argc, char const *argv[])
{
	int i, j;
	float *a = NULL,*b = NULL, *c = NULL;
	//Note: float* a,b,c; does not work, each pointer has to be defined separately
	float *devA, *devB, *devC;	//Pointers in DEVICE

	//CUDA status
	hipError_t cudaStat;
	hipblasStatus_t  stat;
	hipblasHandle_t handle;

	a = (float *)malloc(N*M*sizeof(float));	//N*M matrix
	b = (float *)malloc(N*N*sizeof(float));	//N*N matrix
	c = (float *)malloc(N*M*sizeof(float));	//N*M matrix output
	//Shal be doing b*a, (GEMM)
/*	for (i = 0; i < N; ++i)
	{
		for(j = 0; j<M; ++j)
		{
			*(a + ij2l(i,j,N)) = j + 1;
		}
	}*/

	//Since accesses coalesced when done consecutively, it would be better if we put the coloumn access "j" on the outer loop as j multiplies with N.
	for(j = 0; j<M ; j++)
	{
		for(i = 0;i<N;i++)
		{
			*(a + ij2l(i,j,N)) = j + 1;	//If the older implemetation of ij2l was used, "N" would be "M"
		}
	} 
	for (i = 0; i < N; ++i)
	{
		for (j = 0; j<N; ++j)
		{
			if(i == j)
			{
				*(b + ij2l(i,j,N)) = 0;
			}
			else
			{
				*(b + ij2l(i,j,N)) = 1;
			}
		}
	}
	printf("--------A matrix-----------\n");
	for (i = 0; i < N; ++i)
	{
		for (j = 0; j<M; ++j)
		{
			printf("%f ",*(a + ij2l(i,j,M)) );
		}
		printf("\n");
	}

	printf("--------B matrix-----------\n");
	for (i = 0; i < N; ++i)
	{
		for (j = 0; j<N; ++j)
		{
			printf("%f ",*(b + ij2l(i,j,N)) );
		}
		printf("\n");
	}
	cudaStat = hipMalloc ((void**)&devA, M*N*sizeof(*a)); // Whatever *a is, float or int, space is allocated
	if (cudaStat!=hipSuccess)
	{
		printf("Device Memory Allocation Failed\n");
		return EXIT_FAILURE;
	}
	cudaStat = hipMalloc((void**)&devB, N*N*sizeof(*b));
	if (cudaStat!=hipSuccess)
	{
		printf("Device Memory Allocation Failed\n");
		return EXIT_FAILURE;
	}
	cudaStat = hipMalloc((void**)&devC, N*M*sizeof(float));


	stat = hipblasSetMatrix(N,M,sizeof(*a),a,N,devA,N);
	stat = hipblasSetMatrix(N,N,sizeof(*b),b,N,devB,N);


	const float alpha = 1.0f, beta = 0.0f;
	//For some reason, const float alpha = 1f, beta 0f; does not work. .0 has to be added.

	stat = hipblasCreate(&handle);
		if (stat != HIPBLAS_STATUS_SUCCESS) {
	    fprintf(stderr, "!!!! CUBLAS initialization error\n");
	    return EXIT_FAILURE;
	}

	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,N,M,N,&alpha,devB,N,devA,N,&beta,devC,N);	//Answer is copied in C
	//CUBLAS_OP_N implies that it is not transposed, it is neccessary to give this, else doesn't work
	//alpha and beta have to be given in as pointers, use &alpha, &beta;
	stat = hipblasGetMatrix(N,M,sizeof(*c),devC,N,c,N);
	hipFree(devA);
	hipFree(devB);
	hipFree(devC);
	hipblasDestroy(handle);

	//GEMM does alpha*A*B + beta*C. Now A is N * M, B is N * N. SO we would do B*A.

	printf("--------C matrix-----------\n");
	for (i = 0; i < N; ++i)
	{
		for (j = 0; j<M; ++j)
		{
			printf("%f ",*(c + i + j*N) ); //Coloumn First access
		}
		printf("\n");
	}

	free(a);
	free(b);
	free(c);

	return EXIT_SUCCESS;
}

//With help from https://github.com/zchee/cuda-sample/blob/master/0_Simple/matrixMulCUBLAS/matrixMulCUBLAS.cpp