#include "hip/hip_runtime.h"
// %%cuda --name depth_conv.cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include <cstdlib>
#include <hipblas.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#define IDX2C(i,j,ld) (((j)*(ld))+(i))


__global__ void im2col(float *mat, float *col, int K, int channels, int height, int width, int height_col, int width_col, int stride)
{
    
    int tid_j = blockIdx.x*blockDim.x + threadIdx.x;    //column number
    int tid_i = blockIdx.y*blockDim.y + threadIdx.y;    //row number
    int gid = tid_i*(height_col*width_col) + tid_j;    //global_id when reading row major form
    
    if(tid_j < (height_col*width_col))
    {
        int c_im = blockIdx.y;

        int c = gid/(height_col*width_col);//row in which we are working on in the o/p matrix 
        
        int h_offset = (c/K)%K;
        int w_offset = c%K;
        int h =  (gid%(height_col*width_col))/width_col;
        int w = gid%width_col;


        
        int h_pad = h*stride + h_offset;
        int w_pad = w*stride + w_offset;
        
        int index = (c_im * height + h_pad) * width + w_pad;
        
        col[gid] = mat[index];
            
    }
}


__global__ void rearrange_weights(float* wt_mat, float* out_wt_mat, int K, int channels)
{
    int gid = blockIdx.x*blockDim.x + threadIdx.x;
    if(gid < channels*K*K)
    {
      int row = gid/(K*K);  //the row in the final output matrix that this thread has to work on 
      int off_set = row*(K*K*channels) + row*(K*K) + gid%(K*K); //Exact position where we have to put the value
      out_wt_mat[off_set] = wt_mat[gid];
      
    }
}


void gpuCublasMmul(float *A,  float *B, float *reference,  int m,  int k,  int n) {
    //int lda=m,ldb=k,ldc=m;
    //A = m*k, B = k*n, C = m*n
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;
    // do the actual multiplication
    
    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,n,m,k,alpha,B,n,A,k,beta,reference,n);    
}

void depth_conv(float *d_mat, float * d_wt_mat, float **out_mat, int stride, int channels, int K, int height, int width, float* im2col_time, float* diag_time, float* cublas_time)
{
    int width_col = (width- K)/stride + 1;
    int height_col = (height - K)/stride + 1;
    size_t totalThreads = channels*K*K*height_col*width_col;            //total elements im2col operation
    size_t dim1 = channels*K*K;                                         //size of weight matrix
    size_t dim2 = channels*channels*K*K;                                //size of output weight matrix
    size_t size = channels*height*width;

    //mat is copied to d_mat and d_mat is used ahead.
    hipError_t error = hipSuccess;
 

    if(d_mat == NULL)
    {
        fprintf(stderr, "depth_convDriver: Input Matrix memory not allocated\n");
        exit(EXIT_FAILURE);       
    }

    // float* d_mat = NULL;
    // error = hipMalloc((void **)&d_mat, size*sizeof(float));
    // if(error != hipSuccess) {
    //     fprintf(stderr,"Some Error in hipMalloc for d_mat %s\n",hipGetErrorString(error));
    //     exit(EXIT_FAILURE);
    // }
 
    // hipMemcpy(d_mat, mat, size*sizeof(float), hipMemcpyHostToDevice);
    
    float* d_col = NULL;

    error = hipMalloc((void **)&d_col, totalThreads*sizeof(float));
    if(error != hipSuccess) {
        fprintf(stderr,"depth_convDriver: hipMalloc for d_col %s\n",hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    
    hipDeviceProp_t devp;
    hipGetDeviceProperties(&devp, 0);
    // printf("Warp Size: %d\n", devp.warpSize);
    // printf("Max number of threads per block: %d\n", devp.maxThreadsPerBlock);

    float num_th = 128.0;
    dim3 gridWeightDim(ceil((channels*K*K)/num_th), 1, 1);
    dim3 blockWeightDim(num_th, 1, 1);
 
      dim3 gridDim(ceil((height_col*width_col)/32.0), channels, 1);
    dim3 blockDim(32, K*K, 1);
 
    float* d_wt_mat = NULL;
    error = hipMalloc((void **)&d_wt_mat, dim1*sizeof(float));
    if(error != hipSuccess) {
        fprintf(stderr,"depth_convDriver: Some Error in hipMalloc for d_wt_mat %s\n",hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    
    //weights are in a single matrix, row major. 
    //Assumed to be available in system host RAM. Changed to device
    // hipMemcpy(d_wt_mat, weights, dim1*sizeof(float), hipMemcpyHostToDevice);
    if (d_wt_mat == NULL)
    {
        fprintf(stderr, "depth_convDriver: No Kernel Paramaters Provided\n");
    }
    float* d_out_wt_mat = NULL;
    //float* out_wt_mat = (float *)calloc(dim2, sizeof(float));
    error = hipMalloc((void **)&d_out_wt_mat, dim2*sizeof(float));
    if(error != hipSuccess) {
        fprintf(stderr,"depth_convDriver: Some Error in hipMalloc for d_out_wt_mat %s\n",hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    
    //hipMemcpy(d_out_wt_mat, out_wt_mat, dim2*sizeof(float), hipMemcpyHostToDevice);

    //printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n",
    //gridWeightDim.x, gridWeightDim.y, gridWeightDim.z, blockWeightDim.x, blockWeightDim.y, blockWeightDim.z);

    hipEvent_t start3, stop3;
    float milliseconds3 = 0;
    hipEventCreate( & start3);
    hipEventCreate( & stop3);
    hipEventRecord(start3);

    rearrange_weights<<<gridWeightDim, blockWeightDim>>>(d_wt_mat, d_out_wt_mat, K, channels);

    hipEventRecord(stop3);
    hipEventSynchronize(stop3);
    hipEventElapsedTime( & milliseconds3, start3, stop3);
    printf("Weight Diagonalization: The elapsed time in GPU was %f ms\n", milliseconds3);
    *diag_time = milliseconds3;


    //float* rearranged_weights = (float *)malloc(dim2*sizeof(float));
    //hipMemcpy(rearranged_weights, d_out_wt_mat, dim2*sizeof(float), hipMemcpyDeviceToHost);
    
    //printf("Output rearrangement:\n");
    /*for (int i = 0; i < (channels); ++i)
    {
        for (int j = 0; j < (channels*K*K); ++j)
        {
            printf("%1.1f ", rearranged_weights[i*(channels*K*K) + j]);
        }
        printf("\n");
    }*/
    //hipMemcpy(d_out_wt_mat, rearranged_weights, dim2*sizeof(float), hipMemcpyHostToDevice);
    
    hipEvent_t start1, stop1;
    float milliseconds1 = 0;
    hipEventCreate( & start1);
    hipEventCreate( & stop1);
    hipEventRecord(start1);

    im2col<<<gridDim ,blockDim>>>(d_mat, d_col, K, channels, height, width, height_col, width_col, stride);
    
    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    hipEventElapsedTime( & milliseconds1, start1, stop1);
    printf("Im2Col : The elapsed time in GPU was %f ms\n", milliseconds1);
    *im2col_time = milliseconds1;
    /*float* col_mat = (float *)malloc(totalThreads*sizeof(float));
    hipMemcpy(col_mat, d_col, (totalThreads)*sizeof(float), hipMemcpyDeviceToHost);

    printf("Printing after im2col operation\n");
    for (int i = 0; i < (channels*K*K); ++i)
    {
      for (int j = 0; j < (height_col*width_col); ++j)
      {
        printf("%1.1f ",col_mat[i*(height_col*width_col) + j]);
      }
      printf("\n");
    }
 
    hipMemcpy(d_col, col_mat, totalThreads*sizeof(float), hipMemcpyHostToDevice);
    */
    //printf("0\n");
    float* d_out_mat = NULL;
    error = hipMalloc((void **)&d_out_mat, channels*width_col*height_col*sizeof(float));        
    if(error != hipSuccess) {
        fprintf(stderr,"depth_convDriver: Error in hipMalloc for Output Matrix (d_out_mat) %s\n",hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    
    //printf("2\n");
    int nr_rows_A = channels;
    int nr_cols_A = channels*K*K;
    int nr_cols_B = height_col*width_col;
    
    hipEvent_t start2, stop2;
    float milliseconds2 = 0;
    hipEventCreate( & start2);
    hipEventCreate( & stop2);
    hipEventRecord(start2);


    gpuCublasMmul(d_out_wt_mat, d_col, d_out_mat, nr_rows_A, nr_cols_A, nr_cols_B);

    hipEventRecord(stop2);
    hipEventSynchronize(stop2);
    hipEventElapsedTime( & milliseconds2, start2, stop2);
    printf("cuBLAS : The elapsed time in GPU was %f ms\n", milliseconds2);
    *cublas_time = milliseconds2;

    // hipMemcpy(out_mat, d_out_mat, channels*width_col*height_col*sizeof(float), hipMemcpyDeviceToHost);
    
    // hipFree(d_out_mat);
    // hipFree(d_mat); //It is the duty of the input kernel to free the memory.
    hipFree(d_col);
    // hipFree(d_wt_mat);
    hipFree(d_out_wt_mat);
    // hipFree(d_out_mat);
    *out_mat = d_out_mat;
    //d_out_mat is the output matrix allocated in the memory
    
    //free(rearranged_weights);
    //free(col_mat);
}

int main()
{
    int K, height, width, stride, channels; //kernel size , height of image, width of image, stride, number of channels in the input
    printf("Enter kernel size , height of image, width of image, stride, number of channels in the input\n");
    
    //K SHOULD NOT BE LARGER THAN 5, NOT NEEDED IN THIS ARCHITECTURE ANYWAY. OURS IS CONSTRAINED BY BLOCK DIMENSIONS
    // 6*6*32 > 1024
    hipError_t error = hipSuccess;

    float im2_col_total = 0;
    float diag_total = 0;
    float cublas_total = 0;
 
    float im2col_time = 0;
    float diag_time = 0;
    float cublas_time = 0;

    scanf("%d",&K);
    scanf("%d",&height);
    scanf("%d",&width);
    scanf("%d",&stride);
    scanf("%d",&channels);
    /* height = 50; width = 50; channels = 12;
    stride = 1;channels = 8;K = 3;height = 1024; width = 1024; */
    
    int group_size = 8;//number of channels in a group 
    //int num = ceil(channels/group_size);
    //printf("num = %d\n",num);
    int width_col = (width- K)/stride + 1;
    int height_col = (height - K)/stride + 1;
    
    float* wt_mat = (float *)malloc((channels*K*K)*sizeof(float));
    for(int i = 0; i < channels*K*K; i ++)
    {
          wt_mat[i] = 1;
    }
    
    /*
    printf("Weight Matrix \n");
    for(int i = 0; i < channels; i++)
    {
        for(int j = 0; j < K; j++)
      {
        for(int k = 0; k < K;k++)
        {
            printf("%1.1f ",wt_mat[i*K*K + j*K + k]);
        }
            printf("\n");
      }
      printf("\n");
    }
    */
 
      size_t size = channels*height*width;
    float* input_mat = (float *)malloc(size*sizeof(float));
    
    float* d_input_mat = NULL;
    error = hipMalloc((void **)&d_input_mat, size*sizeof(float));        
    if(error != hipSuccess) {
        fprintf(stderr,"depth_convExample: Error in hipMalloc for Input Matrix %s\n",hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    for(int i = 0; i < size; i++)
    {
            input_mat[i] = 1;
    }
 
    error = hipMemcpy(d_input_mat, input_mat, size*sizeof(float), hipMemcpyHostToDevice);
    if(error != hipSuccess) {
        fprintf(stderr,"depth_convExample: Error in copying input matrix to Device %s\n",hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }


    
    float* out_mat = (float *)malloc(channels*height_col*width_col*sizeof(float));
    float* d_out_mat = NULL;    

    //depth_conv(handle1, input_mat, wt_mat, out_mat, stride, channels, K,  height, width);
    
    int input_offset;
    int weight_offset;
    int output_offset;
    int current_channels = group_size;
    
    hipEvent_t start, stop;
    float milliseconds = 0;

    hipEventCreate( & start);
    hipEventCreate( & stop);

    hipEventRecord(start);
    printf("Entering\n");

    for(int i = 0; i < channels; i+= group_size)
    {
        input_offset = height*width*i;
      weight_offset = K*K*i;
      output_offset = height_col*width_col*i;
      if ((channels - i) < group_size)
        current_channels = channels - i;
      
      depth_conv(input_mat+input_offset, wt_mat+weight_offset, out_mat+output_offset , stride, current_channels, K,  height, width, &im2col_time, &diag_time, &cublas_time);
      im2_col_total += im2col_time;
      diag_total += diag_time;
      cublas_total += cublas_time;
    }
    printf("Left\n");
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime( & milliseconds, start, stop);


    printf("Depthwise Conv: The elapsed time in GPU was %f ms\n", milliseconds);
    printf("Im2Col: The elapsed time in GPU was %f ms\n", im2_col_total);
    printf("Diagonalwise: The elapsed time in GPU was %f ms\n", diag_total);
    printf("CuBlas GEMM: The elapsed time in GPU was %f ms\n", cublas_total);    

    /*printf("Printing the input image\n");
    for (int i = 0; i < channels; ++i)
    {
      for (int j = 0; j < height; ++j)
      {
        for (int k = 0; k < width; ++k)
        {
          printf("%1.1f ",input_mat[i*height*width + j*width + k]);
        }
        printf("\n");
      }
      printf("\n");
    }
    
    printf("Printing the output matrix\n");
    for(int i = 0; i < channels; i++)
    {
        for(int j = 0; j < height_col*width_col; j++)
      {
        printf("%1.1f ", out_mat[i*height_col*width_col + j]);
      }
      printf("\n");
    }*/
    

    free(wt_mat);
    free(input_mat);
    free(out_mat);
    
}