#include "hip/hip_runtime.h"
%%cuda --name gemm.cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <time.h>
#define TileSize 32
    
//Optimised GEMM implementation using shared memory / Tiling method.

__global__ void gemm2(float *A, float *B,float *C,int m,int n,int k)
{
    
    //Calculating index of tile where data is to loaded
    int r=threadIdx.x;
    int c=threadIdx.y;

    //Calculating index of that element in input matrix
    int globalRow=TileSize*blockIdx.x + r;
    int globalCol=TileSize*blockIdx.y + c;

    //Shared location to load tiles
    __shared__ float S1[TileSize][TileSize];
    __shared__ float S2[TileSize][TileSize];

    //Using val as final ans for this index and number of tiles to be iterated per thread
    float val=0;
    int numTiles=ceil((float)((float)k)/((float)TileSize));

    //Bringing in each tile values
    for(int i=0;i<numTiles;i++)
    {
        //Calculating the index from where the value needs to be loaded
        int row=TileSize*i + r;
        int col=TileSize*i + c;

        if(col<k && globalRow<m)
        S1[c][r]=A[col*m + globalRow];
        else S1[c][r]=0;

        if(row<k && globalCol<n)
        S2[c][r]=B[globalCol*k + row];
        else S2[c][r]=0;

        __syncthreads();

        //Mulltiplying the two tiles and adding their sum
        for(int j=0;j<TileSize;j++)
        {
            val+=S1[j][r]*S2[c][j];
        }

        __syncthreads();
        

    }

    //Storing the value in the output array
    if(globalCol<n && globalRow<m)
    C[globalCol*m + globalRow]=val;
}

int main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int length1,length2,width1,width2;

    // Input the length and width of the matrices
    printf("Enter values of length1,width1,length2,width2:");
    scanf("%d %d %d %d",&length1,&width1,&length2,&width2);

    //Calculate size of matrices in bytes
    
    int numElements = length1*width1;
    size_t size = numElements * sizeof(float);

    int numElements2 = length2*width2;
    size_t size2 = numElements2 * sizeof(float);

    int numElements3 = length1*width2;
    size_t size3 = numElements3 * sizeof(float);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size2);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size3);


    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C==NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors for testing
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i]=i%8;
    }

    for(int i=0;i<numElements2;i++)
    {
        h_B[i]=i%8;
    }
    
    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size2);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size3);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

   
    // Copy the host input vectors A and B in host memory to the device input vectors in device memory
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size2, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Launch the GEMM CUDA Kernel

    //Define block and grid dimensions
    dim3 block(32,32,1);
    dim3 grid(max(length1,length2)/32 + 1,max(width1,width2)/32 + 1,1);

    //Use cuda events to determine time taken
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    //Launch the GEMM kernel
    gemm2<<<grid, block>>>(d_A, d_B, d_C,length1,width2,width1);

    //Calculate the time taken by the Kernel
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("The elapsed time in gpu was %f ms", milliseconds);

    //Check for any error in launch of kernel
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch gemm kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector in host memory.
    err = hipMemcpy(h_C, d_C, size3 , hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }



     // Verify that the result vector is correct by performing the operation in CPU

    float arr[length1][width2];//Result array made in CPU side to verify the results

    for(int i=0;i<length1;i++){
        for(int j=0;j<width2;j++){
            arr[i][j]=0;
        }
    }

    //Monitor Time taken in serial execution in CPU side for comparison
    clock_t cpu_start, cpu_end;
    double cpu_time_used;
 
    cpu_start = clock();

    int f=0;
    for(int i=0;i<length1;i++)
    {
        for(int j=0;j<width2;j++)
        {
            for(int k=0;k<length2;k++)
            {
                arr[i][j]+=h_A[k*length1 + i]*h_B[j*length2 + k];
            }

            if(arr[i][j]!=h_C[j*length1 + i]){
                f=1;
            }
        }
    }

    /*
      Code to print both side results if necessary

      for(int i=0;i<length1;i++)
      {
          for(int j=0;j<width2;j++)
          {
              printf("%f ",arr[i][j]);
          }
          printf("\n");
      }
      for(int i=0;i<length1;i++)
      {
          for(int j=0;j<width2;j++)
          {
              printf("%f ",h_C[j*length1 + i]);
          }
          printf("\n");
      }
  */
  
    //Serial time execution printing
    cpu_end = clock();
    cpu_time_used = ((double) (cpu_end - cpu_start)) / CLOCKS_PER_SEC;
    printf("\nTime elapsed in serial execution:%f ms\n",cpu_time_used*1000.00);

    //If both CPU side and GPU side results match or not
    if(!f)
        printf("Success!!\n");
    else
        printf("Failure!!\n");

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    
    // Reset the device and exit
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);
    
    return 0;
}
